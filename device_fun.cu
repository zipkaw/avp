#include "hip/hip_runtime.h"
#include <iostream>
#include <array>
#include "vect_fun.cu"

#define distr_range(random_fun, min, max) (random_fun * (max - (min))) + min

const unsigned int BLOCK_SIZE = 512;

__global__ void block_agregation_kernel(
    unsigned int *global_mem,
    unsigned int *result,
    unsigned int parentGridDim)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    extern __shared__ unsigned int temp[];
    temp[tid] = global_mem[idx];
    
    __syncthreads();
    for (unsigned int s = 1; s < blockDim.x ; s *= 2)
    {
        if (tid % (2 * s) == 0 && (tid + s) < blockDim.x )
        {
            temp[tid] += temp[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        global_mem[0] += temp[0];
        __syncthreads();
        hipMemcpyAsync(result,
                        &global_mem[0],
                        sizeof(unsigned int),
                        hipMemcpyDeviceToDevice);
    }
}

__global__ void volume_tetrahedron_on_device(
    const float3 &A,
    const float3 &B,
    const float3 &C,
    const float3 &D,
    unsigned int N,
    unsigned int *accumulator,
    unsigned int *global_mem)
{
    int tid = threadIdx.x;
    extern __shared__ unsigned int block_count[];
    hiprandState state;
    hiprand_init(clock64(), tid, 0, &state);
    float3 P{static_cast<float>(distr_range(hiprand_uniform(&state), -1.5, 0.3)),
             static_cast<float>(distr_range(hiprand_uniform(&state), -0.2, 0.4)),
             static_cast<float>(distr_range(hiprand_uniform(&state), -0.7, 0.5))};

    if (inside_tetrahedron(A, B, C, D, P))
    {
        atomicAdd(&global_mem[blockIdx.x], 1U);
    }

    __syncthreads();
    if (tid == 0 && blockIdx.x == 0)
    {
        hipStream_t stream;
        hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
        unsigned int childBlockSize = min(gridDim.x, 1024);
        dim3 childBlock{childBlockSize};
        dim3 childGrid{(gridDim.x + childBlockSize -1)/ childBlockSize };
        block_agregation_kernel<<<childGrid, childBlock, gridDim.x / 2 * sizeof(unsigned int), stream>>>(global_mem, accumulator, gridDim.x);
    }
    __syncthreads();
}

unsigned long long device_estimate(
    const std::array<float3, 4> &vertices,
    const unsigned long long &N)
{
    int count = 0;
    int block_size = BLOCK_SIZE;
    int grid_size = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    unsigned int *accumulator;
    hipMalloc(&accumulator, sizeof(unsigned int));
    hipMemset(accumulator, 0, sizeof(unsigned int));

    unsigned int *global_mem;
    hipMallocManaged(&global_mem, sizeof(unsigned int) * grid_size);
    hipMemset(global_mem, 0, sizeof(unsigned int));

    float3 *dev_vertices;
    hipMallocManaged(&dev_vertices, sizeof(float3) * 4);
    hipMemcpy(dev_vertices, vertices.data(), sizeof(float3) * 4, hipMemcpyHostToDevice);

    volume_tetrahedron_on_device<<<grid_size, block_size, sizeof(unsigned int)>>>(
        dev_vertices[0],
        dev_vertices[1],
        dev_vertices[2],
        dev_vertices[3],
        N,
        accumulator,
        global_mem);

    hipMemcpy(&count, accumulator, sizeof(unsigned int), hipMemcpyDeviceToHost);

    return count;
}